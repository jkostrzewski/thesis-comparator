#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>


__global__ void kernel(char *dbText, char *pattern, char *result, int patternLength)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	bool flag = false;
	for (int j = 0; j < patternLength; j++){
		if (dbText[i + j] != pattern[j]){
			flag = true;
			break;
		}
	}
	if (flag){
		result[i] = 0;
	}
	else{
		result[i] = 1;
	}
}

int main(){
	return 0;
}
